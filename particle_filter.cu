#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <float.h>
#include <stdio.h>
#define _USE_MATH_DEFINES
#include <math.h>
#include <hiprand/hiprand_kernel.h>

#define THREADSIZE 4
#define BLOCKSIZE 4

__device__ void propagate(
    int index, float* X1_prev, float* X2_prev, const float* L, float dt) 
{

    float g = 9.81;
    hiprandState state;
    hiprand_init(index, 0, 0, &state);

    float w1 = hiprand_normal(&state);
    float w2 = hiprand_normal(&state);

    X1_prev[index] = X1_prev[index] + X2_prev[index] * dt + w1 * L[0] + w2 * L[2];
    X2_prev[index] = X2_prev[index] - g * sin(X1_prev[index]) * dt + w1 * L[1] + w2 * L[3];
}

__device__ void log_likehood_pendulum(
    int index, float* x1, float* W, float y, float stdev, int N)
{

    float var = stdev * stdev; // sigma^2

    float log_likehood = -0.5 * N * log(2 * M_PI) - 0.5 * N * log(var);
    float diff = y - sin(x1[index]);
    log_likehood += -0.5 / var * (diff * diff);
    W[index] = log_likehood;

}

__device__ void max_ws(int index, float* ws, float* result, int size) {

    extern __shared__ float shared_ws[THREADSIZE];
    int tx = threadIdx.x;
    shared_ws[tx] = (index < size) ? ws[index] : -FLT_MAX;
    __syncthreads();

    for (int i = blockDim.x / 2; i > 0; i >>= 1) {

        if (tx < i) {
            shared_ws[tx] = (shared_ws[tx] < shared_ws[tx + i]) ? shared_ws[tx + i] : shared_ws[tx];
        }
        __syncthreads();


    }

    if (threadIdx.x == 0)
        result[blockIdx.x] = shared_ws[0];
}


__device__ void max_value(
    int index, float* W, int N, float* result)
{
    int size = N;
    while (size > 1) {
        max_ws(index, W, result, size);
        size = ceil(float(size) / THREADSIZE);
    }
}


__global__ void particleFilter(float* X1, float* X2, float dt, int N,int J, float* L,float* x1_prev, float* x2_prev, float* W, float* y, float* max_val, float stdev) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    for (int n = 0; n < N; n++) {
        int matrix_ind = index + n * N;

        propagate(index, x1_prev, x2_prev, L, dt);
        __syncthreads();
        log_likehood_pendulum(index, x1_prev, W, y[n], stdev, N);
        __syncthreads();
        max_value(index, W, N, max_val);
        __syncthreads();
        X1[matrix_ind] = max_val[0];
        X2[matrix_ind] = max_val[0];
        
        
	}
}